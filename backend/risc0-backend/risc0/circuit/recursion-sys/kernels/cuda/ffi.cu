// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "hip/hip_runtime.h"
#include "fp.h"
#include "fpext.h"
#include "kernels.h"

using namespace risc0::circuit::recursion;

extern "C" {

const char* risc0_circuit_recursion_cuda_eval_check(Fp* check,
                                                    const Fp* ctrl,
                                                    const Fp* data,
                                                    const Fp* accum,
                                                    const Fp* mix,
                                                    const Fp* out,
                                                    const Fp& rou,
                                                    uint32_t po2,
                                                    uint32_t domain,
                                                    const FpExt* poly_mix_pows) {
  hipMemcpyToSymbol(HIP_SYMBOL(poly_mix), poly_mix_pows, sizeof(poly_mix));
  return launchKernel(eval_check, domain, 0, check, ctrl, data, accum, mix, out, rou, po2, domain);
}

const char* risc0_circuit_recursion_cuda_step_compute_accum(
    const Fp* ctrl, const Fp* data, const Fp* mix, FpExt* wom, uint32_t steps, uint32_t count) {
  return launchKernel(step_compute_accum, count, 0, ctrl, data, mix, wom, steps, count);
}

const char* risc0_circuit_recursion_cuda_step_verify_accum(const Fp* ctrl,
                                                           const Fp* data,
                                                           const Fp* mix,
                                                           FpExt* wom,
                                                           Fp* accum,
                                                           uint32_t steps,
                                                           uint32_t count) {
  return launchKernel(step_verify_accum, count, 0, ctrl, data, mix, wom, accum, steps, count);
}

} // extern "C"
