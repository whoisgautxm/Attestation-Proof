#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "context.h"
#include "hip/hip_runtime.h"
#include "fp.h"
#include "fpext.h"
#include "kernels.h"

#include "vendor/nvtx3/nvtx3.hpp"

#include <cstring>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

constexpr size_t kStepModeSeqParallel = 0;
constexpr size_t kStepModeSeqForward = 1;
constexpr size_t kStepModeSeqReverse = 2;

namespace {

using CodeReg = size_t;
using OutReg = size_t;
using DataReg = size_t;
using MixReg = size_t;
using AccumReg = size_t;

#include "../../../rv32im/src/layout.cu.inc"

} // namespace

// constexpr size_t kVerifyMemBodyKind = 1;
constexpr size_t kVerifyMemHaltKind = 2;

struct HostContext {
  MachineContext* ctx;

  HostContext(PreflightTrace* trace, size_t steps) {
    CUDA_OK(hipMallocManaged(&ctx, sizeof(MachineContext)));
    ctx->steps = steps;

    CUDA_OK(hipMallocManaged(&ctx->trace, sizeof(PreflightTrace)));
    ctx->trace->isTrace = trace->isTrace;

    ctx->trace->numCycles = trace->numCycles;
    // printf("numCycles: %u\n", trace->numCycles);
    CUDA_OK(hipMalloc(&ctx->trace->cycles, trace->numCycles * sizeof(PreflightCycle)));
    CUDA_OK(hipMemcpy(ctx->trace->cycles,
                       trace->cycles,
                       trace->numCycles * sizeof(PreflightCycle),
                       hipMemcpyHostToDevice));

    ctx->trace->numTxns = trace->numTxns;
    // printf("numTxns: %u\n", trace->numTxns);
    CUDA_OK(hipMalloc(&ctx->trace->txns, trace->numTxns * sizeof(MemoryTransaction)));
    CUDA_OK(hipMemcpy(ctx->trace->txns,
                       trace->txns,
                       trace->numTxns * sizeof(MemoryTransaction),
                       hipMemcpyHostToDevice));

    ctx->trace->numExtras = trace->numExtras;
    // printf("numExtras: %u\n", trace->numExtras);
    CUDA_OK(hipMalloc(&ctx->trace->extras, trace->numExtras * sizeof(uint32_t)));
    CUDA_OK(hipMemcpy(ctx->trace->extras,
                       trace->extras,
                       trace->numExtras * sizeof(uint32_t),
                       hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&ctx->ramRows, steps * kMaxRamRowsPerCycle * sizeof(RamArgumentRow)));
    CUDA_OK(hipMemset(
        ctx->ramRows, kInvalidPattern, steps * kMaxRamRowsPerCycle * sizeof(RamArgumentRow)));

    CUDA_OK(hipMalloc(&ctx->ramIndex, steps * sizeof(uint32_t)));
    CUDA_OK(hipMemset(ctx->ramIndex, 0, steps * sizeof(uint32_t)));

    CUDA_OK(hipMalloc(&ctx->pairs, steps * kMaxBytePairsPerCycle * sizeof(uint32_t)));
    CUDA_OK(
        hipMemset(ctx->pairs, kInvalidPattern, steps * kMaxBytePairsPerCycle * sizeof(uint32_t)));

    CUDA_OK(hipMalloc(&ctx->pairsIndex, steps * sizeof(uint32_t)));
    CUDA_OK(hipMemset(ctx->pairsIndex, 0, steps * sizeof(uint32_t)));
  }

  ~HostContext() {
    // printf("~HostContext\n");
    hipFree(ctx->trace->cycles);
    hipFree(ctx->trace->txns);
    hipFree(ctx->trace->extras);
    hipFree(ctx->trace);
    hipFree(ctx->ramRows);
    hipFree(ctx->ramIndex);
    hipFree(ctx->pairs);
    hipFree(ctx->pairsIndex);
    hipFree(ctx);
  }
};

__device__ bool MachineContext::isParSafeExec(uint32_t cycle) const {
  return trace->cycles[cycle].isSafeExec;
}

__device__ uint8_t MachineContext::isParSafeVerifyMem(uint32_t cycle) const {
  return trace->cycles[cycle].isSafeVerifyMem;
}

struct StepExec {
  __host__ __device__ static const char* name() { return "step_exec"; }

  __device__ static bool is_safe(MachineContext* ctx, uint32_t cycle) {
    return ctx->isParSafeExec(cycle);
  }

  __device__ static void
  step(MachineContext* ctx, uint32_t steps, uint32_t cycle, Fp* arg0, Fp* arg1, Fp* arg2) {
    step_exec(ctx, steps, cycle, arg0, arg1, arg2, nullptr, nullptr);
  }
};

struct StepVerifyMem {
  __host__ __device__ static const char* name() { return "step_verify_mem"; }

  __device__ static bool is_safe(MachineContext* ctx, uint32_t cycle) {
    return ctx->isParSafeVerifyMem(cycle);
  }

  __device__ static void
  step(MachineContext* ctx, uint32_t steps, uint32_t cycle, Fp* arg0, Fp* arg1, Fp* arg2) {
    step_verify_mem(ctx, steps, cycle, arg0, arg1, arg2, nullptr, nullptr);
  }
};

struct StepVerifyBytes {
  __host__ __device__ static const char* name() { return "step_verify_bytes"; }

  __device__ static bool is_safe(MachineContext* ctx, uint32_t cycle) { return true; }

  __device__ static void
  step(MachineContext* ctx, uint32_t steps, uint32_t cycle, Fp* arg0, Fp* arg1, Fp* arg2) {
    step_verify_bytes(ctx, steps, cycle, arg0, arg1, arg2, nullptr, nullptr);
  }
};

template <typename Stage>
__device__ void next_step(MachineContext* ctx,
                          uint32_t steps,
                          uint32_t count,
                          uint32_t cycle,
                          Fp* arg0,
                          Fp* arg1,
                          Fp* arg2) {
  if (cycle == 0 || Stage::is_safe(ctx, cycle)) {
    // printf("%s(%u)\n", Stage::name(), cycle);
    Stage::step(ctx, steps, cycle++, arg0, arg1, arg2);
    while (cycle < count && !Stage::is_safe(ctx, cycle)) {
      // printf("next, %s(%u)\n", Stage::name(), cycle);
      Stage::step(ctx, steps, cycle++, arg0, arg1, arg2);
    }
  }
}

template <typename Stage>
__global__ void
par_step(MachineContext* ctx, uint32_t steps, uint32_t count, Fp* arg0, Fp* arg1, Fp* arg2) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }
  next_step<Stage>(ctx, steps, count, cycle, arg0, arg1, arg2);
}

template <typename Stage>
__global__ void
fwd_step(MachineContext* ctx, uint32_t steps, uint32_t count, Fp* arg0, Fp* arg1, Fp* arg2) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  if (cycle == 0) {
    while (cycle < count) {
      Stage::step(ctx, steps, cycle++, arg0, arg1, arg2);
    }
  }
}

template <typename Stage>
__global__ void
rev_step(MachineContext* ctx, uint32_t steps, uint32_t count, Fp* arg0, Fp* arg1, Fp* arg2) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  if (cycle == count - 1) {
    for (uint32_t i = 0; i < count; i++) {
      uint32_t cycle = count - i - 1;
      next_step<Stage>(ctx, steps, count, cycle, arg0, arg1, arg2);
    }
  }
}

void MachineContext::sortRam() {
  // printf("sortRam\n");
  nvtx3::scoped_range range("sortRam");
  {
    nvtx3::scoped_range range("sort");
    thrust::sort(thrust::device, ramRows, ramRows + steps * kMaxRamRowsPerCycle);
  }

  {
    thrust::host_vector<RamArgumentRow> h_ramRows(steps * kMaxRamRowsPerCycle);

    nvtx3::scoped_range range("dirty");
    CUDA_OK(hipMemcpy(h_ramRows.data(),
                       ramRows,
                       h_ramRows.size() * sizeof(RamArgumentRow),
                       hipMemcpyDeviceToHost));

    uint32_t prevDirty = 0;
    for (size_t i = 0; i < steps * kMaxRamRowsPerCycle; i++) {
      RamArgumentRow& row = h_ramRows[i];
      switch (row.getMemOp()) {
      case 0: // pageIo
        row.dirty = 0;
        break;
      case 1: // read
        row.dirty = prevDirty;
        break;
      case 2: // write
        row.dirty = 1;
        break;
      }
      prevDirty = row.dirty;
    }

    CUDA_OK(hipMemcpy(ramRows,
                       h_ramRows.data(),
                       h_ramRows.size() * sizeof(RamArgumentRow),
                       hipMemcpyHostToDevice));
  }

  {
    nvtx3::scoped_range range("scan");
    thrust::exclusive_scan(thrust::device, ramIndex, ramIndex + steps, ramIndex);
  }
}

__global__ void inject_backs_ram(MachineContext* ctx, uint32_t steps, uint32_t count, Fp* data) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  uint8_t kind = ctx->isParSafeVerifyMem(cycle);
  if (cycle > 2 && kind) {
    uint32_t idx = ctx->ramIndex[cycle];
    assert(idx != 0);

    const RamArgumentRow& back1 = ctx->ramRows[idx - 1];
    constexpr auto header = kLayout.mux.body.header;
    constexpr auto a = header.element;
    constexpr auto v = header.verifier;
    data[a.addr * steps + cycle - 1] = back1.addr;                 // a->addr
    data[a.cycle * steps + cycle - 1] = back1.getMemCycle();       // a->cycle
    data[a.memOp * steps + cycle - 1] = back1.getMemOp();          // a->memOp
    data[a.data[0] * steps + cycle - 1] = back1.word & 0xff;       // a->data[0]
    data[a.data[1] * steps + cycle - 1] = back1.word >> 8 & 0xff;  // a->data[1]
    data[a.data[2] * steps + cycle - 1] = back1.word >> 16 & 0xff; // a->data[2]
    data[a.data[3] * steps + cycle - 1] = back1.word >> 24 & 0xff; // a->data[3]
    data[v.dirty * steps + cycle - 1] = back1.dirty;               // prevVerifier->dirty
    if (kind == kVerifyMemHaltKind) {
      const RamArgumentRow& back2 = ctx->ramRows[idx - 2];
      uint32_t isNewAddr = back2.addr != back1.addr;
      uint32_t cmp;
      if (isNewAddr) {
        cmp = back1.addr - back2.addr - 1;
      } else {
        cmp =
            back1.getMemCycle() * 3 + back1.getMemOp() - back2.getMemCycle() * 3 + back2.getMemOp();
      }
      uint32_t diff[3];
      for (size_t i = 0; i < 3; i++) {
        diff[i] = cmp & 0xff;
        cmp = cmp >> 8;
      }
      uint32_t extra = cmp;
      data[v.isNewAddr * steps + cycle - 1] = isNewAddr; // isNewAddr
      data[v.diff[0] * steps + cycle - 1] = diff[0];     // diff[0]
      data[v.diff[1] * steps + cycle - 1] = diff[1];     // diff[1]
      data[v.diff[2] * steps + cycle - 1] = diff[2];     // diff[2]
      data[v.extra * steps + cycle - 1] = extra;         // extra
    }
  }
}

void MachineContext::sortBytes() {
  nvtx3::scoped_range range("sortBytes");

  {
    nvtx3::scoped_range range("sort");
    thrust::sort(thrust::device, pairs, pairs + steps * kMaxBytePairsPerCycle);
  }

  {
    nvtx3::scoped_range range("scan");
    thrust::exclusive_scan(thrust::device, pairsIndex, pairsIndex + steps, pairsIndex);
  }
}

__global__ void inject_backs_bytes(MachineContext* ctx, size_t steps, size_t count, Fp* data) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle == 0 || cycle >= count) {
    return;
  }

  uint32_t idx = ctx->pairsIndex[cycle];
  uint32_t pair;
  if (idx) {
    pair = ctx->pairs[idx - 1];
  } else {
    pair = 0;
  }
  // printf("inject_backs_bytes[%u]> 0x%x\n", cycle, pair);
  data[0 * steps + cycle - 1] = pair >> 8 & 0xff;
  data[1 * steps + cycle - 1] = pair & 0xff;
}

template <typename Stage>
void run_stage(CudaStream& stream,
               LaunchConfig& cfg,
               MachineContext* ctx,
               uint32_t mode,
               uint32_t last_cycle,
               Fp* ctrl,
               Fp* io,
               Fp* data) {
  // printf("%s\n", stage);
  nvtx3::scoped_range range(Stage::name());
  switch (mode) {
  case kStepModeSeqParallel: {
    par_step<Stage>
        <<<cfg.grid, cfg.block, 0, stream>>>(ctx, ctx->steps, last_cycle, ctrl, io, data);
    CUDA_OK(hipStreamSynchronize(stream));
  } break;
  case kStepModeSeqForward: {
    fwd_step<Stage>
        <<<cfg.grid, cfg.block, 0, stream>>>(ctx, ctx->steps, last_cycle, ctrl, io, data);
    CUDA_OK(hipStreamSynchronize(stream));
  } break;
  case kStepModeSeqReverse: {
    rev_step<Stage>
        <<<cfg.grid, cfg.block, 0, stream>>>(ctx, ctx->steps, last_cycle, ctrl, io, data);
    CUDA_OK(hipStreamSynchronize(stream));
  } break;
  }
}

extern "C" {

const char* risc0_circuit_rv32im_cuda_witgen(uint32_t mode,
                                             PreflightTrace* trace,
                                             uint32_t steps,
                                             uint32_t last_cycle,
                                             Fp* ctrl,
                                             Fp* io,
                                             Fp* data) {
  try {
    nvtx3::scoped_range range("witgen");

    // printf("risc0_circuit_rv32im_cuda_witgen\n");
    CUDA_OK(hipDeviceSynchronize());

    HostContext ctx(trace, steps);

    CudaStream stream;
    LaunchConfig cfg = getSimpleConfig(last_cycle);

    run_stage<StepExec>(stream, cfg, ctx.ctx, mode, last_cycle, ctrl, io, data);

    {
      nvtx3::scoped_range range("verify_ram");
      ctx.ctx->sortRam();

      {
        // printf("inject_backs_ram\n");
        nvtx3::scoped_range range("inject_backs_ram");
        inject_backs_ram<<<cfg.grid, cfg.block, 0, stream>>>(ctx.ctx, steps, last_cycle, data);
        CUDA_OK(hipStreamSynchronize(stream));
      }

      run_stage<StepVerifyMem>(stream, cfg, ctx.ctx, mode, last_cycle, ctrl, io, data);
    }

    {
      nvtx3::scoped_range range("verify_bytes");
      ctx.ctx->sortBytes();

      {
        // printf("inject_backs_bytes\n");
        nvtx3::scoped_range range("inject_backs_bytes");
        inject_backs_bytes<<<cfg.grid, cfg.block, 0, stream>>>(ctx.ctx, steps, last_cycle, data);
        CUDA_OK(hipStreamSynchronize(stream));
      }

      run_stage<StepVerifyBytes>(stream, cfg, ctx.ctx, mode, last_cycle, ctrl, io, data);
    }
  } catch (const std::runtime_error& err) {
    return strdup(err.what());
  }
  return nullptr;
}

__global__ void par_step_compute_accum(AccumContext* ctx,
                                       uint32_t steps,
                                       uint32_t count,
                                       Fp* arg0,
                                       Fp* arg1,
                                       Fp* arg2,
                                       Fp* arg3,
                                       Fp* arg4) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }
  step_compute_accum(ctx, steps, cycle, arg0, arg1, arg2, arg3, arg4);
}

const char* risc0_circuit_rv32im_cuda_step_compute_accum(AccumContext* ctx,
                                                         uint32_t steps,
                                                         uint32_t count,
                                                         Fp* ctrl,
                                                         Fp* io,
                                                         Fp* data,
                                                         Fp* mix,
                                                         Fp* accum) {
  try {
    CUDA_OK(hipDeviceSynchronize());

    CudaStream stream;
    auto cfg = getSimpleConfig(count);

    par_step_compute_accum<<<cfg.grid, cfg.block, 0, stream>>>(
        ctx, steps, count, ctrl, io, data, mix, accum);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::runtime_error& err) {
    return strdup(err.what());
  }
  return nullptr;
}

__global__ void par_step_verify_accum(AccumContext* ctx,
                                      uint32_t steps,
                                      uint32_t count,
                                      Fp* arg0,
                                      Fp* arg1,
                                      Fp* arg2,
                                      Fp* arg3,
                                      Fp* arg4) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }
  step_verify_accum(ctx, steps, cycle, arg0, arg1, arg2, arg3, arg4);
}

const char* risc0_circuit_rv32im_cuda_step_verify_accum(AccumContext* ctx,
                                                        uint32_t steps,
                                                        uint32_t count,
                                                        Fp* ctrl,
                                                        Fp* io,
                                                        Fp* data,
                                                        Fp* mix,
                                                        Fp* accum) {
  try {
    CUDA_OK(hipDeviceSynchronize());

    CudaStream stream;
    auto cfg = getSimpleConfig(count);
    par_step_verify_accum<<<cfg.grid, cfg.block, 0, stream>>>(
        ctx, steps, count, ctrl, io, data, mix, accum);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::runtime_error& err) {
    return strdup(err.what());
  }
  return nullptr;
}

const char* risc0_circuit_rv32im_cuda_eval_check(Fp* check,
                                                 const Fp* ctrl,
                                                 const Fp* data,
                                                 const Fp* accum,
                                                 const Fp* mix,
                                                 const Fp* out,
                                                 const Fp& rou,
                                                 uint32_t po2,
                                                 uint32_t domain,
                                                 const FpExt* poly_mix_pows) {
  try {
    CUDA_OK(hipDeviceSynchronize());

    CudaStream stream;
    auto cfg = getSimpleConfig(domain);
    hipMemcpyToSymbol(HIP_SYMBOL(poly_mix), poly_mix_pows, sizeof(poly_mix));
    eval_check<<<cfg.grid, cfg.block, 0, stream>>>(
        check, ctrl, data, accum, mix, out, rou, po2, domain);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::runtime_error& err) {
    return strdup(err.what());
  }
  return nullptr;
}

} // extern "C"
