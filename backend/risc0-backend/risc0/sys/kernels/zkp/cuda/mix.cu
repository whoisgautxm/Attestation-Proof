#include "hip/hip_runtime.h"
// Copyright 2022 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"
#include "fpext.h"

__global__ void mix_poly_coeffs(FpExt* out,
                                const Fp* in,
                                const uint32_t* combos,
                                const FpExt* mixStart,
                                const FpExt* mix,
                                const uint32_t inputSize,
                                const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    FpExt cur = *mixStart;
    for (size_t i = 0; i < inputSize; i++) {
      size_t id = combos[i];
      out[count * id + idx] += cur * in[count * i + idx];
      cur *= *mix;
    }
  }
}
