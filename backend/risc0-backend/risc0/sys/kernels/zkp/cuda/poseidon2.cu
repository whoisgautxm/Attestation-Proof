#include "hip/hip_runtime.h"
// Copyright 2023 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"
#include "sha256.h"

#define CELLS 24
#define ROUNDS_FULL 8
#define ROUNDS_HALF_FULL (ROUNDS_FULL / 2)
#define ROUNDS_PARTIAL 21
#define ROW_SIZE (CELLS + ROUNDS_PARTIAL)
#define CELLS_RATE 16
#define CELLS_OUT 8

namespace poseidon2 {

__device__ void add_round_constants_full(const Fp* ROUND_CONSTANTS, Fp* cells, uint round) {
  for (uint i = 0; i < CELLS; i++) {
    cells[i] += ROUND_CONSTANTS[round * CELLS + i];
  }
}

__device__ void add_round_constants_partial(const Fp* ROUND_CONSTANTS, Fp* cells, uint round) {
  cells[0] += ROUND_CONSTANTS[round * CELLS];
}

__device__ Fp sbox(Fp x) {
  Fp x2 = x * x;
  Fp x4 = x2 * x2;
  Fp x6 = x4 * x2;
  return x6 * x;
}

__device__ void do_full_sboxes(Fp* cells) {
  for (uint i = 0; i < CELLS; i++) {
    cells[i] = sbox(cells[i]);
  }
}

__device__ void do_partial_sboxes(Fp* cells) {
  cells[0] = sbox(cells[0]);
}

__device__ void multiply_by_m_int(const Fp* M_INT_DIAG, Fp* cells) {
  // Exploit the fact that off-diagonal entries of M_INT are all 1.
  Fp sum = 0;
  for (uint i = 0; i < CELLS; i++) {
    sum += cells[i];
  }
  for (uint i = 0; i < CELLS; i++) {
    cells[i] = sum + M_INT_DIAG[i] * cells[i];
  }
}

__device__ void multiply_by_4x4_circulant(Fp* x) {
  // See appendix B of Poseidon2 paper.
  Fp t0 = x[0] + x[1];
  Fp t1 = x[2] + x[3];
  Fp t2 = Fp(2) * x[1] + t1;
  Fp t3 = Fp(2) * x[3] + t0;
  Fp t4 = Fp(4) * t1 + t3;
  Fp t5 = Fp(4) * t0 + t2;
  Fp t6 = t3 + t5;
  Fp t7 = t2 + t4;
  x[0] = t6;
  x[1] = t5;
  x[2] = t7;
  x[3] = t4;
}

__device__ void multiply_by_m_ext(Fp* old_cells) {
  // Optimized method for multiplication by M_EXT.
  // See appendix B of Poseidon2 paper for additional details.
  Fp cells[CELLS];
  for (uint i = 0; i < CELLS; i++) {
    cells[0] = 0;
  }
  Fp tmp_sums[4];
  for (uint i = 0; i < 4; i++) {
    tmp_sums[i] = 0;
  }
  for (uint i = 0; i < CELLS / 4; i++) {
    multiply_by_4x4_circulant(old_cells + i * 4);
    for (uint j = 0; j < 4; j++) {
      Fp to_add = old_cells[i * 4 + j];
      tmp_sums[j] += to_add;
      cells[i * 4 + j] += to_add;
    }
  }
  for (uint i = 0; i < CELLS; i++) {
    old_cells[i] = cells[i] + tmp_sums[i % 4];
  }
}

__device__ void full_round(const Fp* ROUND_CONSTANTS, Fp* cells, uint round) {
  add_round_constants_full(ROUND_CONSTANTS, cells, round);
  do_full_sboxes(cells);
  multiply_by_m_ext(cells);
}

__device__ void
partial_round(const Fp* ROUND_CONSTANTS, const Fp* M_INT_DIAG, Fp* cells, uint round) {
  add_round_constants_partial(ROUND_CONSTANTS, cells, round);
  do_partial_sboxes(cells);
  multiply_by_m_int(M_INT_DIAG, cells);
}

__device__ void poseidon2_mix(const Fp* ROUND_CONSTANTS, const Fp* M_INT_DIAG, Fp* cells) {
  uint round = 0;

  // First linear layer.
  multiply_by_m_ext(cells);

  // Do initial full rounds
  for (uint i = 0; i < ROUNDS_HALF_FULL; i++) {
    full_round(ROUND_CONSTANTS, cells, round);
    round++;
  }
  // Do partial rounds
  for (uint i = 0; i < ROUNDS_PARTIAL; i++) {
    partial_round(ROUND_CONSTANTS, M_INT_DIAG, cells, round);
    round++;
  }
  // Do remaining full rounds
  for (uint i = 0; i < ROUNDS_HALF_FULL; i++) {
    full_round(ROUND_CONSTANTS, cells, round);
    round++;
  }
}

} // namespace poseidon2

__global__ void poseidon2_fold(const Fp* ROUND_CONSTANTS,
                               const Fp* M_INT_DIAG,
                               Fp* output,
                               const Fp* input,
                               uint32_t output_size) {
  uint32_t gid = blockDim.x * blockIdx.x + threadIdx.x;
  if (gid >= output_size) {
    return;
  }
  Fp cells[CELLS];
  for (size_t i = 0; i < CELLS_OUT; i++) {
    cells[i] = input[2 * gid * CELLS_OUT + i];
    cells[CELLS_OUT + i] = input[(2 * gid + 1) * CELLS_OUT + i];
  }
  poseidon2::poseidon2_mix(ROUND_CONSTANTS, M_INT_DIAG, cells);
  for (uint i = 0; i < CELLS_OUT; i++) {
    output[gid * CELLS_OUT + i] = cells[i];
  }
}

__global__ void poseidon2_rows(const Fp* ROUND_CONSTANTS,
                               const Fp* M_INT_DIAG,
                               Fp* out,
                               const Fp* matrix,
                               uint32_t count,
                               uint32_t col_size) {
  uint32_t gid = blockDim.x * blockIdx.x + threadIdx.x;
  if (gid >= count) {
    return;
  }
  Fp cells[CELLS];
  uint used = 0;
  for (uint i = 0; i < col_size; i++) {
    cells[used++] = matrix[i * count + gid];
    if (used == CELLS_RATE) {
      poseidon2::poseidon2_mix(ROUND_CONSTANTS, M_INT_DIAG, cells);
      used = 0;
    }
  }
  if (used != 0 || count == 0) {
    // Zero pad to get a CELLS_RATE-aligned number of inputs
    for (uint i = used; i < CELLS_RATE; i++) {
      cells[i] = 0;
    }
    poseidon2::poseidon2_mix(ROUND_CONSTANTS, M_INT_DIAG, cells);
  }
  for (uint i = 0; i < CELLS_OUT; i++) {
    out[CELLS_OUT * gid + i] = cells[i];
  }
}
