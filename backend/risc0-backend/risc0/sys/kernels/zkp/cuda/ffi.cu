// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "hip/hip_runtime.h"
#include "fp.h"
#include "fpext.h"
#include "kernels.h"

extern "C" {

const char* risc0_zkp_cuda_eltwise_add_fp(Fp* out, const Fp* x, const Fp* y, uint32_t count) {
  return launchKernel(eltwise_add_fp, count, 0, out, x, y, count);
}

const char* risc0_zkp_cuda_eltwise_mul_factor_fp(Fp* io, Fp factor, uint32_t count) {
  return launchKernel(eltwise_mul_factor_fp, count, 0, io, factor, count);
}

const char* risc0_zkp_cuda_eltwise_copy_fp(Fp* out, const Fp* in, const uint32_t count) {
  return launchKernel(eltwise_copy_fp, count, 0, out, in, count);
}

const char* risc0_zkp_cuda_eltwise_sum_fpext(Fp* out,
                                             const FpExt* in,
                                             const uint32_t to_add,
                                             const uint32_t count) {
  return launchKernel(eltwise_sum_fpext, count, 0, out, in, to_add, count);
}

const char* risc0_zkp_cuda_eltwise_zeroize_fp(Fp* elems, const uint32_t count) {
  return launchKernel(eltwise_zeroize_fp, count, 0, elems);
}

const char* risc0_zkp_cuda_eltwise_zeroize_fpext(FpExt* elems, const uint32_t count) {
  return launchKernel(eltwise_zeroize_fpext, count, 0, elems);
}

const char* risc0_zkp_cuda_fri_fold(Fp* out, const Fp* in, const FpExt* mix, const uint32_t count) {
  return launchKernel(fri_fold, count, 0, out, in, mix, count);
}

const char* risc0_zkp_cuda_mix_poly_coeffs(FpExt* out,
                                           const Fp* in,
                                           const uint32_t* combos,
                                           const FpExt* mixStart,
                                           const FpExt* mix,
                                           const uint32_t inputSize,
                                           const uint32_t count) {
  return launchKernel(mix_poly_coeffs, count, 0, out, in, combos, mixStart, mix, inputSize, count);
}

const char* risc0_zkp_cuda_batch_bit_reverse(Fp* io, const uint32_t nBits, const uint32_t count) {
  return launchKernel(batch_bit_reverse, count, 0, io, nBits, count);
}

const char* risc0_zkp_cuda_batch_evaluate_any(FpExt* out,
                                              const Fp* coeffs,
                                              const uint32_t* which,
                                              const FpExt* xs,
                                              uint32_t shared_size,
                                              const uint32_t count,
                                              const uint32_t deg) {
  return launchKernel(batch_evaluate_any, count, shared_size, out, coeffs, which, xs, deg);
}

const char* risc0_zkp_cuda_gather_sample(
    Fp* dst, const Fp* src, const uint32_t idx, const uint32_t size, const uint32_t stride) {
  return launchKernel(gather_sample, size, 0, dst, src, idx, size, stride);
}

const char* risc0_zkp_cuda_poseidon_fold(const Fp* round_constants,
                                         const Fp* mds,
                                         const Fp* partial_comp_matrix,
                                         const Fp* partial_comp_offset,
                                         Fp* output,
                                         const Fp* input,
                                         uint32_t output_size) {
  return launchKernel(poseidon_fold,
                      output_size,
                      0,
                      round_constants,
                      mds,
                      partial_comp_matrix,
                      partial_comp_offset,
                      output,
                      input,
                      output_size);
}

const char* risc0_zkp_cuda_poseidon_rows(const Fp* round_constants,
                                         const Fp* mds,
                                         const Fp* partial_comp_matrix,
                                         const Fp* partial_comp_offset,
                                         Fp* output,
                                         const Fp* matrix,
                                         uint32_t row_size,
                                         uint32_t col_size) {
  return launchKernel(poseidon_rows,
                      row_size,
                      0,
                      round_constants,
                      mds,
                      partial_comp_matrix,
                      partial_comp_offset,
                      output,
                      matrix,
                      row_size,
                      col_size);
}

const char* risc0_zkp_cuda_poseidon2_fold(const Fp* round_constants,
                                          const Fp* m_int_diag,
                                          Fp* output,
                                          const Fp* input,
                                          uint32_t output_size) {
  return launchKernel(
      poseidon2_fold, output_size, 0, round_constants, m_int_diag, output, input, output_size);
}

const char* risc0_zkp_cuda_poseidon2_rows(const Fp* round_constants,
                                          const Fp* m_int_diag,
                                          Fp* output,
                                          const Fp* matrix,
                                          uint32_t row_size,
                                          uint32_t col_size) {
  return launchKernel(
      poseidon2_rows, row_size, 0, round_constants, m_int_diag, output, matrix, row_size, col_size);
}

const char*
risc0_zkp_cuda_sha_rows(ShaDigest* output, const Fp* matrix, uint32_t rowSize, uint32_t colSize) {
  return launchKernel(sha_rows, rowSize, 0, output, matrix, rowSize, colSize);
}

const char* risc0_zkp_cuda_sha_fold(ShaDigest* output, const ShaDigest* input, uint32_t count) {
  return launchKernel(sha_fold, count, 0, output, input, count);
}

} // extern "C"
