#include "hip/hip_runtime.h"
// Copyright 2022 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"
#include "fpext.h"

__global__ void batch_bit_reverse(Fp* io, const uint32_t nBits, const uint32_t count) {
  uint totIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (totIdx < count) {
    uint32_t rowSize = 1 << nBits;
    uint32_t idx = totIdx & (rowSize - 1);
    uint32_t s = totIdx >> nBits;
    uint32_t ridx = __brev(idx) >> (32 - nBits);
    if (idx < ridx) {
      size_t idx1 = s * rowSize + idx;
      size_t idx2 = s * rowSize + ridx;
      Fp tmp = io[idx1];
      io[idx1] = io[idx2];
      io[idx2] = tmp;
    }
  }
}

__global__ void batch_evaluate_any(
    FpExt* out, const Fp* coeffs, const uint32_t* which, const FpExt* xs, const uint32_t deg) {
  const Fp* cur_poly = coeffs + which[blockIdx.x] * deg;
  FpExt x = xs[blockIdx.x];
  FpExt stepx = pow(x, blockDim.x);
  FpExt powx = pow(x, threadIdx.x);
  FpExt tot;
  for (size_t i = threadIdx.x; i < deg; i += blockDim.x) {
    tot += powx * cur_poly[i];
    powx *= stepx;
  }
  extern __shared__ uint32_t totsBuf[];
  FpExt* tots = reinterpret_cast<FpExt*>(totsBuf);
  tots[threadIdx.x] = tot;
  __syncthreads();
  unsigned cur = blockDim.x;
  while (cur) {
    cur /= 2;
    if (threadIdx.x < cur) {
      tots[threadIdx.x] = FpExt(tots[threadIdx.x]) + FpExt(tots[threadIdx.x + cur]);
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    out[blockIdx.x] = tots[0];
  }
}

__global__ void gather_sample(
    Fp* dst, const Fp* src, const uint32_t idx, const uint32_t size, const uint32_t stride) {
  uint gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    dst[gid] = src[gid * stride + idx];
  }
}
